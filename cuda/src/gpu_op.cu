#include "hip/hip_runtime.h"
#include "./c_runtime_api.h"
#include <cassert>
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <math.h>

/* TODO: Your code here */
/* all your GPU kernel code, e.g. matrix_softmax_cross_entropy_kernel */

// y = inputs[0], y_ = inputs[1]
// np.mean(-np.sum(y_ * np.log(softmax(y)), axis=1), keepdims=True)
__global__ void matrix_softmax_cross_entropy_kernel(int nrow, int ncol,
                                                    const float *input_a, const float *input_b, float *output) {
    // Dynamic shared memory, size provided at kernel launch.
    extern __shared__ float loss_per_row[];
    // Two dimensional thread blocks.
    int y = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x
            + threadIdx.x;
    if (y >= nrow) {
        return;
    }
    input_a += y * ncol;
    input_b += y * ncol;
    float maxval = *input_a;
    // Find max for a row.
    for (int x = 1; x < ncol; ++x) {
        maxval = max(maxval, input_a[x]);
    }
    // Deduct by max for a row, and raise to exp.
    float sum = 0;
    for (int x = 0; x < ncol; ++x) {
        sum += exp(input_a[x] - maxval);
    }
    // Compute per-row loss.
    float loss = 0;
    for (int x = 0; x < ncol; ++x) {
        loss -= input_b[x] * log(exp(input_a[x] - maxval) / sum);
    }
    loss_per_row[y] = loss;
    __syncthreads();
    // Compute reduce_mean across rows.
    float mean_loss = 0;
    // Use a single thread to reduce mean across rows.
    if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
        for (int i = 0; i < nrow; ++i) {
            mean_loss += loss_per_row[i];
        }
        mean_loss /= nrow;
        output[0] = mean_loss;
    }
}


__global__ void array_set_kernel(float *array, float value, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        array[index] = value;
    }
}


int DLGpuArraySet(DLArrayHandle arr, float value) { /* TODO: Your code here */
    int n = 1;
    for (int i = 0; i < arr->ndim; i++) {
        n = n * arr->shape[i];
    }

    float *array_data = (float *) arr->data;

    int threads_per_block = 1024;
    int num_blocks = (n + threads_per_block - 1) / threads_per_block;

    array_set_kernel << < num_blocks, threads_per_block >> > (array_data, value, n);
    return 0;
}


__global__ void broadcast_to_kernel(const float *input_data,
                                    float *output_data,
                                    index_t input_n,
                                    index_t output_n) {
    index_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < output_n) {
        output_data[idx] = input_data[idx % input_n];
    }
}


int DLGpuBroadcastTo(const DLArrayHandle input, DLArrayHandle output) {
    /* TODO: Your code here */
    index_t input_n = 1;
    for (int i = 0; i < input->ndim; i++)
        input_n *= input->shape[i];

    index_t output_n = 1;
    for (int i = 0; i < output->ndim; i++)
        output_n *= output->shape[i];

    const float *input_data = (const float *) input->data;
    float *output_data = (float *) output->data;

    int thread_per_block = 512;
    int n_blocks = (output_n + thread_per_block - 1) / thread_per_block;
    broadcast_to_kernel << < n_blocks, thread_per_block >> > (input_data, output_data,
            input_n, output_n);
    return 0;
}

__global__ void reduced_sum_axis_zero(const float *input_data, float *output_data, int input_n, int output_n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < output_n) {
        output_data[idx] = 0.0;
        for (int i = 0; i < input_n / output_n; i++) {
            output_data[idx] += input_data[i * output_n + idx];
        }
    }
}

int DLGpuReduceSumAxisZero(const DLArrayHandle input, DLArrayHandle output) {
    /* TODO: Your code here */
    int input_n = 1;
    for (int i = 0; i < input->ndim; i++) {
        input_n *= input->shape[i];
    }

    int output_n = 1;
    for (int i = 0; i < output->ndim; i++) {
        output_n *= output->shape[i];
    }

    const float *input_data = (const float *) input->data;
    float *output_data = (float *) output->data;

    int thread_per_block = 1024;
    int n_blocks = (output_n + thread_per_block - 1) / thread_per_block;

    reduced_sum_axis_zero << < n_blocks, thread_per_block >> > (input_data, output_data, input_n, output_n);
    return 0;
}

__global__ void matrix_elementwise_add(const float *a, const float *b, float *c,
                                       int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

int DLGpuMatrixElementwiseAdd(const DLArrayHandle matA,
                              const DLArrayHandle matB, DLArrayHandle output) {
    /* TODO: Your code here */
    int n = 1;
    for (int i = 0; i < output->ndim; i++) {
        n = n * output->shape[i];
    }
    const float *data_A = (const float *) matA->data;
    const float *data_B = (const float *) matB->data;
    float *data_output = (float *) output->data;

    int threads_per_block = 1024;
    int num_blocks = (n + threads_per_block - 1) / threads_per_block;

    matrix_elementwise_add << < num_blocks, threads_per_block >> > (data_A, data_B,
            data_output, n);
    return 0;
}

__global__
void matrix_elementwise_subtract(const float *a, const float *b, float *c,
                                 int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        c[index] = a[index] - b[index];
    }
}

int DLGpuMatrixElementwiseSubtract(const DLArrayHandle matA,
                                   const DLArrayHandle matB, DLArrayHandle output) {
    /* TODO: Your code here */
    int n = 1;
    for (int i = 0; i < output->ndim; i++) {
        n = n * output->shape[i];
    }
    const float *data_A = (const float *) matA->data;
    const float *data_B = (const float *) matB->data;
    float *data_output = (float *) output->data;

    int threads_per_block = 1024;
    int num_blocks = (n + threads_per_block - 1) / threads_per_block;

    matrix_elementwise_subtract << < num_blocks, threads_per_block >> > (data_A, data_B,
            data_output, n);
    return 0;
}

__global__
void matrix_elementwise_division(const float *a, const float *b, float *result, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        result[index] = a[index] / b[index];
    }
}

int DLGpuMatrixElementwiseDiv(const DLArrayHandle matA, const DLArrayHandle matB,
                              DLArrayHandle output) {
    int n = 1;
    for (int i = 0; i < output->ndim; i++) {
        n = n * output->shape[i];
    }
    const float *data_A = (const float *) matA->data;
    const float *data_B = (const float *) matB->data;
    float *data_output = (float *) output->data;

    int threads_per_block = 1024;
    int num_blocks = (n + threads_per_block - 1) / threads_per_block;

    matrix_elementwise_division << < num_blocks, threads_per_block >> > (data_A, data_B,
            data_output, n);
    return 0;

}

__global__ void matrix_elementwise_add_by_const_kernal(const float *d_in,
                                                       float *d_out, float val, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        d_out[index] = d_in[index] + val;
    }
}

int DLGpuMatrixElementwiseAddByConst(const DLArrayHandle input, float val,
                                     DLArrayHandle output) {
    /* TODO: Your code here */
    int n = 1;
    for (int i = 0; i < output->ndim; i++) {
        n = n * output->shape[i];
    }
    const float *input_data = (const float *) input->data;
    float *output_data = (float *) output->data;
    int threads_per_block = 1024;
    int num_blocks = (n + threads_per_block - 1) / threads_per_block;
    matrix_elementwise_add_by_const_kernal << < num_blocks, threads_per_block >> > (
            input_data, output_data, val, n);
    return 0;
}

__global__
void matrix_elementwise_subtract_by_const_kernal(const float *d_in,
                                                 float *d_out, float val, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        d_out[index] = d_in[index] - val;
    }
}

int DLGpuMatrixElementwiseSubtractByConst(const DLArrayHandle input, float val,
                                          DLArrayHandle output) {
    /* TODO: Your code here */
    int n = 1;
    for (int i = 0; i < output->ndim; i++) {
        n = n * output->shape[i];
    }
    const float *input_data = (const float *) input->data;
    float *output_data = (float *) output->data;
    int threads_per_block = 1024;
    int num_blocks = (n + threads_per_block - 1) / threads_per_block;
    matrix_elementwise_subtract_by_const_kernal << < num_blocks, threads_per_block >> > (
            input_data, output_data, val, n);
    return 0;
}


__global__ void matrix_elementwise_div_by_const_kernal(const float *d_in,
                                                       float *d_out, float val, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        d_out[index] = d_in[index] / val;
    }
}

int DLGpuMatrixElementwiseDivByConst(const DLArrayHandle input, float val,
                                     DLArrayHandle output) {
    /* TODO: Your code here */
    int n = 1;
    for (int i = 0; i < output->ndim; i++) {
        n = n * output->shape[i];
    }
    const float *input_data = (const float *) input->data;
    float *output_data = (float *) output->data;
    int threads_per_block = 1024;
    int num_blocks = (n + threads_per_block - 1) / threads_per_block;
    matrix_elementwise_div_by_const_kernal << < num_blocks, threads_per_block >> > (
            input_data, output_data, val, n);
    return 0;
}


__global__ void elementwise_mul_kernel(const float *data_a, const float *data_b,
                                       float *output, int n) {

    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < n) {
        output[index] = data_a[index] * data_b[index];
    }
}

int DLGpuMatrixElementwiseMultiply(const DLArrayHandle matA,
                                   const DLArrayHandle matB, DLArrayHandle output) {
    /* TODO: Your code here */
    int n = 1;
    for (int i = 0; i < output->ndim; i++) {
        n = n * output->shape[i];
    }

    int threads_per_block = 1024;
    int num_blocks = (n + threads_per_block - 1) / threads_per_block;

    const float *mat_a_data = (const float *) matA->data;
    const float *mat_b_data = (const float *) matB->data;
    float *output_data = (float *) output->data;

    elementwise_mul_kernel << < num_blocks, threads_per_block >> > (mat_a_data,
            mat_b_data, output_data, n);

    return 0;
}

__global__
void matrix_elementwise_sqrt(const float *d_input, float *d_output, int n) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < n) {
        d_output[index] = sqrt(d_input[index]);
    }
}

int DLGpuMatrixElementwiseSqrt(const DLArrayHandle input, DLArrayHandle output) {
    /* TODO: Your code here */
    int n = 1;
    for (int i = 0; i < input->ndim; i++) {
        n *= input->shape[i];
    }

    const float *input_data = (const float *) input->data;
    float *output_data = (float *) output->data;
    int threads_per_block = 1024;
    int num_blocks = (n + threads_per_block - 1) / threads_per_block;
    matrix_elementwise_sqrt << < num_blocks, threads_per_block >> > (input_data, output_data, n);
    return 0;
}


__global__ void marix_multiply_by_const(const float *d_input, float *d_output,
                                        float val, int n) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < n) {
        d_output[index] = d_input[index] * val;
    }
}

int DLGpuMatrixMultiplyByConst(const DLArrayHandle input, float val,
                               DLArrayHandle output) {
    /* TODO: Your code here */
    int n = 1;
    for (int i = 0; i < input->ndim; i++) {
        n *= input->shape[i];
    }

    const float *input_data = (const float *) input->data;
    float *output_data = (float *) output->data;
    int threads_per_block = 1024;
    int num_blocks = (n + threads_per_block - 1) / threads_per_block;
    marix_multiply_by_const << < num_blocks, threads_per_block >> > (input_data,
            output_data, val, n);
    return 0;
}

// int DLGpuMatrixMultiply(const DLArrayHandle matA, bool transposeA,
// 		const DLArrayHandle matB, bool transposeB, DLArrayHandle matC) {
// 	/* TODO: Your code here */
// 	// Hint: use cublas
// 	// cublas assume matrix is column major
//     hipblasHandle_t handle;
//     hipblasStatus_t stat = hipblasCreate(&handle);
//     if (stat != HIPBLAS_STATUS_SUCCESS)
//         printf("CUBLAS initialization failed\n");

//     const float *matA_data = (const float *) matA->data;
//     const float *matB_data = (const float *) matB->data;
//     float *matC_data = (float *) matC->data;

//     hipblasOperation_t transa = transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
//     hipblasOperation_t transb = transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N;

//     int m = transposeB ? matB->shape[0] : matB->shape[1];
//     int n = transposeA ? matA->shape[1] : matA->shape[0];
//     int k = transposeA ? matA->shape[0] : matA->shape[1];

//     float alpha = 1.0f;
//     float beta = 0.0f;
//     stat = hipblasSgemm(handle, transb, transa,
//                        m, n, k,
//                        &alpha, matB_data, matB->shape[1],
//                        matA_data, matA->shape[1],
//                        &beta, matC_data, m);

//     if (stat != HIPBLAS_STATUS_SUCCESS)
//         printf("CUBLAS kernel execution error.\n");

//     stat = hipblasDestroy(handle);
//     if (stat != HIPBLAS_STATUS_SUCCESS)
//         printf("CUBLAS shutdown error\n");

//     return 0;
// }
hipblasHandle_t cublas_handle = NULL;

int DLGpuMatrixMultiply(const DLArrayHandle matA, bool transposeA,
                        const DLArrayHandle matB, bool transposeB,
                        DLArrayHandle matC) {
    /* TODO: Your code here */
    // Hint: use cublas
    // cublas assume matrix is column major
    // op(A) * op(B) = C
    // op(B)T * op(A)T = CT

    if (!cublas_handle) {
        hipblasCreate(&cublas_handle);
    }

    float one = 1.0f;
    float zero = 0.0f;
    int m = matC->shape[1];
    int n = matC->shape[0];
    int k = transposeA ? matA->shape[0] : matA->shape[1];

    hipblasSgemm(cublas_handle,
                transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N,
                m, n, k,
                &one,
                (const float *) matB->data, !transposeB ? m : k,
                (const float *) matA->data, !transposeA ? k : n,
                &zero,
                (float *) matC->data, m
    );
    return 0;
}

__global__ void relu_kernel(const float *input, float *output, int n) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < n) {
        float element = input[index];
        if (element <= 0) {
            output[index] = 0;
        } else {
            output[index] = element;
        }
    }
}

int DLGpuRelu(const DLArrayHandle input, DLArrayHandle output) {
    /* TODO: Your code here */
    int n = 1;
    for (int i = 0; i < input->ndim; i++) {
        n *= input->shape[i];
    }

    const float *input_data = (const float *) input->data;
    float *output_data = (float *) output->data;
    int threads_per_block = 1024;
    int num_blocks = (n + threads_per_block - 1) / threads_per_block;
    relu_kernel << < num_blocks, threads_per_block >> > (input_data, output_data, n);
    return 0;
}

__global__ void relu_gradient_kernel(const float *input, float *output,
                                     const float *in_grad, int n) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < n) {
        float element = input[index];
        if (element <= 0) {
            output[index] = 0;
        } else {
            output[index] = in_grad[index];
        }
    }
}

int DLGpuReluGradient(const DLArrayHandle input, const DLArrayHandle in_grad,
                      DLArrayHandle output) {
    /* TODO: Your code here */
    int n = 1;
    for (int i = 0; i < input->ndim; i++) {
        n *= input->shape[i];
    }

    const float *input_data = (const float *) input->data;
    float *output_data = (float *) output->data;
    const float *in_grad_data = (const float *) in_grad->data;
    int threads_per_block = 1024;
    int num_blocks = (n + threads_per_block - 1) / threads_per_block;

    relu_gradient_kernel << < num_blocks, threads_per_block >> > (input_data,
            output_data, in_grad_data, n);
    return 0;
}

__global__ void softmax_kernel(int64_t nrow, int64_t ncol,
                               const float *input_data,
                               float *output_data) {

// two dimensional thread blocks.
    int y = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    if (y >= nrow) {
        return;
    }
    // y_th row of input data
    input_data += y * ncol;
    output_data += y * ncol;
    // find max for a row.
    float maxval = *input_data;
    for (int x = 1; x < ncol; ++x) {
        maxval = max(maxval, input_data[x]);
    }
    // Deduct by max for a row, and raise to exp.
    // in case of too large of exp, and the result will not be affected
    float sum = 0;
    for (int x = 0; x < ncol; ++x) {
        sum += exp(input_data[x] - maxval);
    }
    // Compute per-row softmax.
    for (int x = 0; x < ncol; ++x) {
        output_data[x] = exp(input_data[x] - maxval) / sum;
    }
}


int DLGpuSoftmax(const DLArrayHandle input, DLArrayHandle output) {
    /* TODO: Your code here */
    assert(input->ndim == 2);
    assert(output->ndim == 2);
    int64_t nrow = input->shape[0];
    int64_t ncol = input->shape[1];
    float *input_data = (float *) input->data;
    float *output_data = (float *) output->data;
    dim3 threads;
    if (nrow < 1024) {
        threads.x = nrow;
    } else {
        threads.x = 1024;
        threads.y = (nrow + 1023) / 1024;
    }
    softmax_kernel << < 1, threads >> > (nrow, ncol, input_data, output_data);
    return 0;
}

int DLGpuSoftmaxCrossEntropy(const DLArrayHandle input_a,
                             const DLArrayHandle input_b, DLArrayHandle output) {
    assert(input_a->ndim == 2);
    assert(input_b->ndim == 2);
    assert(output->ndim == 1);
    assert(
            input_a->shape[0] == input_b->shape[0]
            && input_a->shape[1] == input_b->shape[1]);
    int nrow = input_a->shape[0];
    // Maximum x- or y-dimension of a block = 1024
    // But we need 'nrow' shared memory, and max shared memory is 48KB.
    // Conservatively allow max 16KB shared memory.
    assert(nrow <= 1024 * 4);
    int ncol = input_a->shape[1];
    const float *input_data_a = (const float *) input_a->data;
    const float *input_data_b = (const float *) input_b->data;
    float *output_data = (float *) output->data;
    dim3 threads;
    if (nrow <= 1024) {
        threads.x = nrow;
    } else {
        threads.x = 1024;
        threads.y = (nrow + 1023) / 1024;
    }
    // 1 block, each block with 'threads' number of threads with 'nrow' shared
    // memory size
    matrix_softmax_cross_entropy_kernel << < 1, threads, nrow * sizeof(float) >> > (
            nrow, ncol, input_data_a, input_data_b, output_data);
    return 0;
}